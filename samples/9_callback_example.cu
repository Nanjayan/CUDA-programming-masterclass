#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""
#include "common.h"
#include "cuda_common.cuh"

void CUDART_CB my_callback(hipStream_t stream, hipError_t status, void *data)
{
	printf("callback from stream %d\n", *((int *)data));
}

__global__ void cb_k1()
{
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	if (gid == 0)
	{
		printf("This is a test 1 \n");
	}
}

__global__ void cb_k2()
{
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	if (gid == 0)
	{
		printf("This is a test 2 \n");
	}
}

__global__ void cb_k3()
{
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	if (gid == 0)
	{
		printf("This is a test 3 \n");
	}
}

//int main(int argc, char ** argv)
//{
//	int size = 1 << 15;
//	hipEvent_t event_str1;
//	gpuErrchk(hipEventCreateWithFlags(&event_str1, hipEventDisableTiming));
//
//	hipStream_t stm1, stm2, stm3;
//	gpuErrchk(hipStreamCreate(&stm1));
//	gpuErrchk(hipStreamCreate(&stm2));
//	gpuErrchk(hipStreamCreate(&stm3));
//
//	dim3 block(128);
//	dim3 grid(size / block.x);
//
//	cb_k1 << <grid, block, 0, stm1 >> > ();
//	cb_k2 << <grid, block, 0, stm2 >> > ();
//	cb_k3  << <grid, block, 0, stm3 >> > ();
//
//	gpuErrchk(hipStreamDestroy(stm1));
//	gpuErrchk(hipStreamDestroy(stm2));
//	gpuErrchk(hipStreamDestroy(stm3));
//	gpuErrchk(hipDeviceSynchronize());
//
//	gpuErrchk(hipDeviceReset());
//	return 0;
//}
