#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""

#include "cuda_common.cuh"

__global__ void atomics(int *shared_var, int iters)
{
	for (int i = 0; i < iters; i++)
	{
		atomicAdd(shared_var, 1);
	}
}

__global__ void unsafe(int *shared_var, int iters)
{
	for (int i = 0; i < iters; i++)
	{
		int old = *shared_var;
		*shared_var = old + 1;
	}
}

//int main(int argc, char **argv)
//{
//	int N = 64;
//	int block = 32;
//	int runs = 30;
//	int iters = 100000;
//	int r;
//	int *d_shared_var;
//	int h_shared_var_atomic, h_shared_var_unsafe;
//	int *h_values_read;
//
//	gpuErrchk(hipMalloc((void **)&d_shared_var, sizeof(int)));
//
//	double atomic_mean_time = 0;
//	double unsafe_mean_time = 0;
//	clock_t ops_start, ops_end;
//
//	for (r = 0; r < runs; r++)
//	{
//		gpuErrchk(hipMemset(d_shared_var, 0x00, sizeof(int)));
//	
//		ops_start = clock();
//		atomics <<< N / block, block >>>(d_shared_var,iters);
//		gpuErrchk(hipDeviceSynchronize());
//		ops_end = clock();
//		atomic_mean_time += ops_end - ops_start;
//
//		gpuErrchk(hipMemcpy(&h_shared_var_atomic, d_shared_var, sizeof(int),
//			hipMemcpyDeviceToHost));
//		gpuErrchk(hipMemset(d_shared_var, 0x00, sizeof(int)));
//		
//		ops_start = clock();
//		unsafe <<< N / block, block >>>(d_shared_var,iters);
//		gpuErrchk(hipDeviceSynchronize());
//		ops_end = clock();
//		unsafe_mean_time += ops_end - ops_start;
//		
//		gpuErrchk(hipMemcpy(&h_shared_var_unsafe, d_shared_var, sizeof(int),
//			hipMemcpyDeviceToHost));
//	}
//
//	atomic_mean_time = atomic_mean_time / CLOCKS_PER_SEC;
//	unsafe_mean_time = unsafe_mean_time / CLOCKS_PER_SEC;
//
//	printf("In total, %d runs using atomic operations took %f s\n",
//		runs, atomic_mean_time);
//	printf("  Using atomic operations also produced an output of %d\n",
//		h_shared_var_atomic);
//	printf("In total, %d runs using unsafe operations took %f s\n",
//		runs, unsafe_mean_time);
//	printf("  Using unsafe operations also produced an output of %d\n",
//		h_shared_var_unsafe);
//
//	return 0;
//}
