
#include <hip/hip_runtime.h>
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//
//#include <stdio.h>
//#include <stdlib.h>
//
//__global__ void unique_index_calc_threadIdx(int * data)
//{
//	int tid = threadIdx.x;
//	printf("threadIdx.x : %d - data : %d \n", tid, data[tid]);
//}
//
//
//__global__ void unique_gid_calculation(int * data)
//{
//	int tid = threadIdx.x;
//	int offset = blockIdx.x * blockDim.x;
//	int gid = tid + offset;
//
//	printf("blockIdx.x : %d, threadIdx.x : %d - data : %d \n",
//		blockIdx.x, tid, data[gid]);
//}
//
//int main()
//{
//	int array_size = 16;
//	int array_byte_size = sizeof(int) * array_size;
//	int h_data[] = { 23,9,4,53,65,12,1,33,22,43,56,1,76,81,94,32 };
//
//	for (int i = 0; i < array_size; i++)
//	{
//		printf("%d ", h_data[i]);
//	}
//	printf("\n \n");
//
//	int * d_data;
//	cudaMalloc((void**)&d_data, array_byte_size);
//	cudaMemcpy(d_data, h_data, array_byte_size, cudaMemcpyHostToDevice);
//
//	dim3 block(4);
//	dim3 grid(2);
//
//	unique_index_calc_threadIdx << < grid, block >> > (d_data);
//	cudaDeviceSynchronize();
//
//	cudaDeviceReset();
//	return 0;
//}