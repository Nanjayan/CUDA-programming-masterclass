#include <stdio.h>
#include <stdlib.h>
#include <math.h>

#include "hip/hip_runtime.h"


__global__ void standard(float *ptr)
{
	*ptr = powf(*ptr, 2.0f);
}

__global__ void intrinsic(float *ptr)
{ 
	*ptr = __powf(*ptr, 2.0f);
}

//int main()
//{
//	float value = 23;	
//	int SIZE = sizeof(float);
//
//	float *d_val;
//	cudaMalloc((void**)&d_val, SIZE);
//	cudaMemcpy(d_val, &value, SIZE, cudaMemcpyHostToDevice);
//	standard << <1, 1 >> > (d_val);
//	intrinsic << <1, 1 >> > (d_val);
//	cudaDeviceSynchronize();
//
//	cudaDeviceReset();
//	return 0;
//}