#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""

#include "common.h"

__global__ void stream_test_modified(int* in, int * out, int size)
{
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	if (gid < size)
	{
		//THIS FOR LOOP IS ONLY FOR VISUALIZING PURPOSE  
		for (int i = 0; i < 25; i++)
		{
			out[gid] = in[gid] + (in[gid] - 1) * (gid % 10);
		}
	}
}

//int main(int argc, char ** argv)
//{
//	int size = 1 << 18;
//	int byte_size = size * sizeof(int);
//
//	//initialize host pointer
//	int* h_in, *h_ref, *h_in2, *h_ref2;
//	
//	hipHostMalloc((void**)&h_in,byte_size);
//	hipHostMalloc((void**)&h_ref, byte_size);
//	hipHostMalloc((void**)&h_in2, byte_size);
//	hipHostMalloc((void**)&h_ref2, byte_size);
//
//	initialize(h_in, INIT_RANDOM);
//	initialize(h_in2, INIT_RANDOM);
//
//
//	//allocate device pointers
//	int * d_in, *d_out, *d_in2, *d_out2;
//	hipMalloc((void**)&d_in, byte_size);
//	hipMalloc((void**)&d_out, byte_size);
//	hipMalloc((void**)&d_in2, byte_size);
//	hipMalloc((void**)&d_out2, byte_size);
//
//	hipStream_t str,str2;
//	hipStreamCreate(&str);
//	hipStreamCreate(&str2);
//
//
//	//kernel launch
//	dim3 block(128);
//	dim3 grid(size / block.x);
//	
//	//transfer data from host to device
//	hipMemcpyAsync(d_in, h_in, byte_size, hipMemcpyHostToDevice,str);
//	stream_test_modified << <grid, block,0,str >> > (d_in, d_out, size);
//	hipMemcpyAsync(h_ref, d_out, byte_size, hipMemcpyDeviceToHost,str);
//	
//	hipMemcpyAsync(d_in2, h_in2, byte_size, hipMemcpyHostToDevice, str2);
//	stream_test_modified << <grid, block, 0, str2 >> > (d_in2, d_out2, size);
//	hipMemcpyAsync(h_ref2, d_out2, byte_size, hipMemcpyDeviceToHost, str2);
//	
//	hipStreamSynchronize(str);
//	hipStreamDestroy(str);
//
//	hipStreamSynchronize(str2);
//	hipStreamDestroy(str2);
//
//	hipDeviceReset();
//	return 0;
//}