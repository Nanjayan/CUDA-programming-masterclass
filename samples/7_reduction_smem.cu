#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "common.h"
#include "cuda_common.cuh"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#define BLOCK_SIZE 1024

template<unsigned int iblock_size>
__global__ void reduction_gmem_benchmark(int * input,int * temp, int size)
{
	int tid = threadIdx.x;
	int * i_data = input + blockDim.x * blockIdx.x;

	//manual unrolling depending on block size
	if (iblock_size >= 1024 && tid < 512)
		i_data[tid] += i_data[tid + 512];

	__syncthreads();

	if (iblock_size >= 512 && tid < 256)
		i_data[tid] += i_data[tid + 256];

	__syncthreads();

	if (iblock_size >= 256 && tid < 128)
		i_data[tid] += i_data[tid + 128];

	__syncthreads();

	if (iblock_size >= 128 && tid < 64)
		i_data[tid] += i_data[tid + 64];

	__syncthreads();

	//unrolling warp
	if (tid < 32)
	{
		volatile int * vsmem = i_data;
		vsmem[tid] += vsmem[tid + 32];
		vsmem[tid] += vsmem[tid + 16];
		vsmem[tid] += vsmem[tid + 8];
		vsmem[tid] += vsmem[tid + 4];
		vsmem[tid] += vsmem[tid + 2];
		vsmem[tid] += vsmem[tid + 1];
	}

	if (tid == 0)
	{
		temp[blockIdx.x] = i_data[0];
	}
}

template<unsigned int iblock_size>
__global__ void reduction_smem(int * input, int * temp, int size)
{
	__shared__ int smem[BLOCK_SIZE];
	int tid = threadIdx.x;
	int * i_data = input + blockDim.x * blockIdx.x;

	smem[tid] = i_data[tid];

	__syncthreads();

	//manual unrolling depending on block size
	if (iblock_size >= 1024 && tid < 512)
		i_data[tid] += i_data[tid + 512];

	__syncthreads();

	if (iblock_size >= 512 && tid < 256)
		i_data[tid] += i_data[tid + 256];

	__syncthreads();

	if (iblock_size >= 256 && tid < 128)
		i_data[tid] += i_data[tid + 128];

	__syncthreads();

	if (iblock_size >= 128 && tid < 64)
		i_data[tid] += i_data[tid + 64];

	__syncthreads();

	//unrolling warp
	if (tid < 32)
	{
		volatile int * vsmem = smem;
		vsmem[tid] += vsmem[tid + 32];
		vsmem[tid] += vsmem[tid + 16];
		vsmem[tid] += vsmem[tid + 8];
		vsmem[tid] += vsmem[tid + 4];
		vsmem[tid] += vsmem[tid + 2];
		vsmem[tid] += vsmem[tid + 1];
	}

	if (tid == 0)
	{
		temp[blockIdx.x] = i_data[0];
	}
}


//int main(int argc, char ** argv)
//{
//    printf("Running parallel reduction with complete unrolling kernel \n");
//
//	int kernel_index = 0;
//
//	if (argc >1)
//	{
//		kernel_index = 1;
//	}
//
//	int size = 1 << 22;
//	int byte_size = size * sizeof(int);
//	int block_size = BLOCK_SIZE;
//
//	int * h_input, *h_ref;
//	h_input = (int*)malloc(byte_size);
//
//	initialize(h_input, size);
//
//	int cpu_result = reduction_cpu(h_input, size);
//
//	dim3 block(block_size);
//	dim3 grid((size / block_size));
//
//	int temp_array_byte_size = sizeof(int)* grid.x;
//
//	h_ref = (int*)malloc(temp_array_byte_size);
//
//	int * d_input, *d_temp;
//	gpuErrchk(hipMalloc((void**)&d_input, byte_size));
//	gpuErrchk(hipMalloc((void**)&d_temp, temp_array_byte_size));
//
//	gpuErrchk(hipMemset(d_temp, 0, temp_array_byte_size));
//	gpuErrchk(hipMemcpy(d_input, h_input, byte_size,
//		hipMemcpyHostToDevice));
//	if (kernel_index == 0)
//	{
//		printf("Kernel launch parameters || grid : %d, block : %d \n", grid.x, block.x);
//
//		switch (block_size)
//		{
//		case 1024:
//			reduction_gmem_benchmark <1024> << < grid, block >> > (d_input, d_temp, size);
//			break;
//		case 512:
//			reduction_gmem_benchmark <512> << < grid, block >> > (d_input, d_temp, size);
//			break;
//		case 256:
//			reduction_gmem_benchmark <256> << < grid, block >> > (d_input, d_temp, size);
//			break;
//		case 128:
//			reduction_gmem_benchmark <128> << < grid, block >> > (d_input, d_temp, size);
//			break;
//		case 64:
//			reduction_gmem_benchmark <64> << < grid, block >> > (d_input, d_temp, size);
//			break;
//		}
//	}
//	else
//	{
//		printf("Kernel launch parameters || grid : %d, block : %d \n", grid.x, block.x);
//
//		switch (block_size)
//		{
//		case 1024:
//			reduction_smem <1024> << < grid, block >> > (d_input, d_temp, size);
//			break;
//		case 512:
//			reduction_smem <512> << < grid, block >> > (d_input, d_temp, size);
//			break;
//		case 256:
//			reduction_smem <256> << < grid, block >> > (d_input, d_temp, size);
//			break;
//		case 128:
//			reduction_smem <128> << < grid, block >> > (d_input, d_temp, size);
//			break;
//		case 64:
//			reduction_smem <64> << < grid, block >> > (d_input, d_temp, size);
//			break;
//		}
//	}
//
//	gpuErrchk(hipDeviceSynchronize());
//	gpuErrchk(hipMemcpy(h_ref, d_temp, temp_array_byte_size, hipMemcpyDeviceToHost));
//
//	int gpu_result = 0;
//	for (int i = 0; i < grid.x; i++)
//	{
//		gpu_result += h_ref[i];
//	}
//
//	compare_results(gpu_result, cpu_result);
//
//	gpuErrchk(hipFree(d_input));
//	gpuErrchk(hipFree(d_temp));
//	free(h_input);
//	free(h_ref);
//
//	gpuErrchk(hipDeviceReset());
//	return 0;
//}