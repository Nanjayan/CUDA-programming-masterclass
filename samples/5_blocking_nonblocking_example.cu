#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""
#include "common.h"
#include "cuda_common.cuh"

__global__ void blocking_nonblocking_test1()
{
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	if (gid == 0)
	{
		for (size_t i = 0; i < 10000; i++)
		{
			printf("kernel 1 \n");
		}
	}
}

//int main(int argc, char ** argv)
//{
//	int size = 1 << 15;
//	
//	hipStream_t stm1,stm2,stm3;
//	gpuErrchk(hipStreamCreateWithFlags(&stm1, hipStreamNonBlocking));
//	gpuErrchk(hipStreamCreate(&stm2));
//	gpuErrchk(hipStreamCreateWithFlags(&stm3,hipStreamNonBlocking));
//
//
//	dim3 block(128);
//	dim3 grid(size / block.x);
//
//	blocking_nonblocking_test1 << <grid, block, 0 , stm1 >> > ();
//	blocking_nonblocking_test1 << <grid, block >> > ();
//	blocking_nonblocking_test1 << <grid, block, 0, stm3 >> > ();
//
//	gpuErrchk(hipStreamDestroy(stm1));
//	gpuErrchk(hipStreamDestroy(stm2));
//	gpuErrchk(hipStreamDestroy(stm3));
//	gpuErrchk(hipDeviceSynchronize());
//
//	gpuErrchk(hipDeviceReset());
//	return 0;
//}