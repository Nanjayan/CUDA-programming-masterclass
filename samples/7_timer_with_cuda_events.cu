#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""
#include "common.h"
#include "cuda_common.cuh"

__global__ void event_test()
{
	double sum = 0.0;
	for (int i = 0; i < 1000; i++)
	{
		sum = sum + tan(0.1) * tan(0.1);
	}
}

//int main(int argc, char ** argv)
//{
//	int size = 1 << 12;
//
//	dim3 block(128);
//	dim3 grid(size / block.x);
//
//	hipEvent_t start, end;
//
//	hipEventCreate(&start);
//	hipEventCreate(&end);
//
//	hipEventRecord(start);
//
//	event_test << < grid,block >>> ();
//
//	hipEventRecord(end);
//	hipEventSynchronize(end);
//
//	float time;
//	hipEventElapsedTime(&time, start, end);
//
//	printf("Kernel execution time using events : %f \n",time);
//
//	hipEventDestroy(start);
//	hipEventDestroy(end);
//
//	hipDeviceReset();
//	return 0;
//}