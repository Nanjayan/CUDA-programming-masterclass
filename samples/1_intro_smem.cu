#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""

#include "common.h"

#define SHARED_ARRAY_SIZE 128

__global__ void smem_static_test(int * in, int * out, int size)
{
	int tid = threadIdx.x;
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	__shared__ int smem[SHARED_ARRAY_SIZE];

	if (gid < size)
	{
		smem[tid] = in[gid];
		out[gid] = smem[tid];
	}
}

__global__ void smem_dynamic_test(int * in, int * out, int size)
{
	int tid = threadIdx.x;
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	extern __shared__ int smem[];

	if (gid < size)
	{
		smem[tid] = in[gid];
		out[gid] = smem[tid];
	}
}

//int main(int argc, char ** argv)
//{
//	int size = 1 << 22;
//	int block_size = SHARED_ARRAY_SIZE;
//	bool dynamic = false;
//
//	if (argc > 1)
//	{
//		dynamic = atoi(argv[1]);
//	}
//
//	//number of bytes needed to hold element count
//	size_t NO_BYTES = size * sizeof(int);
//
//	// host pointers
//	int *h_in, *h_ref, *d_in, *d_out;
//
//	//allocate memory for host size pointers
//	h_in = (int *)malloc(NO_BYTES);
//	h_ref = (int *)malloc(NO_BYTES);
//
//	initialize(h_in, size, INIT_ONE_TO_TEN);
//
//	hipMalloc((int **)&d_in, NO_BYTES);
//	hipMalloc((int **)&d_out, NO_BYTES);
//
//	//kernel launch parameters
//	dim3 block(block_size);
//	dim3 grid((size / block.x) + 1);
//
//	hipMemcpy(d_in, h_in, NO_BYTES, hipMemcpyHostToDevice);
//
//	if (!dynamic)
//	{
//		printf("Static smem kernel \n");
//		smem_static_test << <grid, block >> > (d_in, d_out, size);
//	}
//	else
//	{
//		printf("Dynamic smem kernel \n");
//		smem_dynamic_test << <grid, block, sizeof(int)*  SHARED_ARRAY_SIZE >> > (d_in, d_out, size);
//	}
//	hipDeviceSynchronize();
//
//	hipMemcpy(h_ref, d_out, NO_BYTES, hipMemcpyDeviceToHost);
//
//	hipFree(d_in);
//	hipFree(d_out);
//
//	free(h_in);
//	free(h_ref);
//
//	hipDeviceReset();
//	return EXIT_SUCCESS;
//}