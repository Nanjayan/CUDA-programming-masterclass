#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "common.h"
#include "cuda_common.cuh"

#include "hip/hip_runtime.h"
#include ""

__global__ void test_unified_memory(float* a, float* b, float *c, int size)
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x;

	if (gid < size)
		c[gid] = a[gid] + b[gid];
}

//int main(int argc, char** argv)
//{
//	printf("Runing 1D grid \n");
//	int size = 1 << 22;
//	int block_size = 128;
//
//	if (argc > 1)
//		block_size = 1 << atoi(argv[1]);
//
//	printf("Entered block size : %d \n", block_size);
//
//	unsigned int byte_size = size * sizeof(float);
//
//	printf("Input size : %d \n", size);
//
//	float * A, *B, *ref, *C;
//
//	hipMallocManaged((void **)&A, byte_size);
//	hipMallocManaged((void **)&B, byte_size);
//	hipMallocManaged((void **)&ref, byte_size);
//
//	C = (float*)malloc(byte_size);
//
//	if (!A)
//		printf("host memory allocation error \n");
//
//	for (size_t i = 0; i < size; i++)
//	{
//		A[i] = i % 10;
//		A[i] = i % 7;
//	}
//
//	sum_array_cpu(A, B, C, size);
//
//	dim3 block(block_size);
//	dim3 grid((size + block.x - 1) / block.x);
//
//	printf("Kernel is lauch with grid(%d,%d,%d) and block(%d,%d,%d) \n",
//		grid.x, grid.y, grid.z, block.x, block.y, block.z);
//
//	test_unified_memory << <grid, block >> > (A, B, ref, size);
//	gpuErrchk(hipDeviceSynchronize());
//
//	compare_arrays(ref, C, size);
//	free(C);
//
//	return 0;
//}
