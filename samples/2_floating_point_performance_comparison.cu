#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""
#include "cuda_common.cuh"

__global__ void lots_of_float_compute(float *inputs, int N, size_t niters,
	float *outputs)
{
	size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t nthreads = gridDim.x * blockDim.x;

	for (; tid < N; tid += nthreads)
	{
		size_t iter;
		float val = inputs[tid];

		for (iter = 0; iter < niters; iter++)
		{
			val = (val + 5.0f) - 101.0f;
			val = (val / 3.0f) + 102.0f;
			val = (val + 1.07f) - 103.0f;
			val = (val / 1.037f) + 104.0f;
			val = (val + 3.00f) - 105.0f;
			val = (val / 0.22f) + 106.0f;
		}

		outputs[tid] = val;
	}
}

__global__ void lots_of_double_compute(double *inputs, int N, size_t niters,
	double *outputs)
{
	size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
	size_t nthreads = gridDim.x * blockDim.x;

	for (; tid < N; tid += nthreads)
	{
		size_t iter;
		double val = inputs[tid];

		for (iter = 0; iter < niters; iter++)
		{
			val = (val + 5.0) - 101.0;
			val = (val / 3.0) + 102.0;
			val = (val + 1.07) - 103.0;
			val = (val / 1.037) + 104.0;
			val = (val + 3.00) - 105.0;
			val = (val / 0.22) + 106.0;
		}

		outputs[tid] = val;
	}
}

static void run_float_test(size_t N, int niters, int blocksPerGrid,int threadsPerBlock,
	long* to_device_clock_cyl,long * kernel_clock_cyl, long* from_device_clock_cyl,
	float *sample, int sampleLength) 
{
	int i;
	float *h_floatInputs, *h_floatOutputs;
	float *d_floatInputs, *d_floatOutputs;

	h_floatInputs = (float *)malloc(sizeof(float) * N);
	h_floatOutputs = (float *)malloc(sizeof(float) * N);
	gpuErrchk(hipMalloc((void **)&d_floatInputs, sizeof(float) * N));
	gpuErrchk(hipMalloc((void **)&d_floatOutputs, sizeof(float) * N));

	for (i = 0; i < N; i++)
	{
		h_floatInputs[i] = (float)i;
	}

	clock_t ops_start, ops_end;

	ops_start = clock();
	gpuErrchk(hipMemcpy(d_floatInputs, h_floatInputs, sizeof(float) * N,hipMemcpyHostToDevice));
	ops_end = clock();
	*to_device_clock_cyl = ops_end - ops_start;

	ops_start = clock();
	lots_of_float_compute << <blocksPerGrid, threadsPerBlock >> >(d_floatInputs,N, niters, d_floatOutputs);
	gpuErrchk(hipDeviceSynchronize());
	ops_end = clock();
	*kernel_clock_cyl = ops_end - ops_start;


	ops_start = clock();
	gpuErrchk(hipMemcpy(h_floatOutputs, d_floatOutputs, sizeof(float) * N,hipMemcpyDeviceToHost));
	ops_end = clock();
	*from_device_clock_cyl = ops_end - ops_start;


	for (i = 0; i < sampleLength; i++)
	{
		sample[i] = h_floatOutputs[i];
	}

	gpuErrchk(hipFree(d_floatInputs));
	gpuErrchk(hipFree(d_floatOutputs));
	free(h_floatInputs);
	free(h_floatOutputs);
}

static void run_double_test(size_t N, int niters, int blocksPerGrid,int threadsPerBlock, 
	long* to_device_clock_cyl,long * kernel_clock_cyl, long* from_device_clock_cyl,
	double *sample, int sampleLength)
{
	int i;
	double *h_doubleInputs, *h_doubleOutputs;
	double *d_doubleInputs, *d_doubleOutputs;

	h_doubleInputs = (double *)malloc(sizeof(double) * N);
	h_doubleOutputs = (double *)malloc(sizeof(double) * N);
	gpuErrchk(hipMalloc((void **)&d_doubleInputs, sizeof(double) * N));
	gpuErrchk(hipMalloc((void **)&d_doubleOutputs, sizeof(double) * N));

	for (i = 0; i < N; i++)
	{
		h_doubleInputs[i] = (double)i;
	}

	clock_t ops_start, ops_end;

	ops_start = clock();
	gpuErrchk(hipMemcpy(d_doubleInputs, h_doubleInputs, sizeof(double) * N,
		hipMemcpyHostToDevice));
	ops_end = clock();
	*to_device_clock_cyl = ops_end - ops_start;

	ops_start = clock();
	lots_of_double_compute << <blocksPerGrid, threadsPerBlock >> >(d_doubleInputs,
		N, niters, d_doubleOutputs);
	gpuErrchk(hipDeviceSynchronize());
	ops_end = clock();
	*kernel_clock_cyl = ops_end - ops_start;

	ops_start = clock();
	gpuErrchk(hipMemcpy(h_doubleOutputs, d_doubleOutputs, sizeof(double) * N,
		hipMemcpyDeviceToHost));
	ops_end = clock();
	*from_device_clock_cyl = ops_end - ops_start;

	for (i = 0; i < sampleLength; i++)
	{
		sample[i] = h_doubleOutputs[i];
	}

	gpuErrchk(hipFree(d_doubleInputs));
	gpuErrchk(hipFree(d_doubleOutputs));
	free(h_doubleInputs);
	free(h_doubleOutputs);
}

//int main(int argc, char **argv)
//{
//	int i;
//	double meanFloatToDeviceTime, meanFloatKernelTime, meanFloatFromDeviceTime;
//	double meanDoubleToDeviceTime, meanDoubleKernelTime,
//		meanDoubleFromDeviceTime;
//	struct hipDeviceProp_t deviceProperties;
//	size_t totalMem, freeMem;
//	float *floatSample;
//	double *doubleSample;
//	int sampleLength = 10;
//	int nRuns = 5;
//	int nKernelIters = 20;
//
//	meanFloatToDeviceTime = meanFloatKernelTime = meanFloatFromDeviceTime = 0.0;
//	meanDoubleToDeviceTime = meanDoubleKernelTime =
//		meanDoubleFromDeviceTime = 0.0;
//
//	gpuErrchk(hipMemGetInfo(&freeMem, &totalMem));
//	gpuErrchk(hipGetDeviceProperties(&deviceProperties, 0));
//
//	size_t N = (freeMem * 0.9 / 2) / sizeof(double);
//	int threadsPerBlock = 256;
//	int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
//
//	if (blocksPerGrid > deviceProperties.maxGridSize[0])
//	{
//		blocksPerGrid = deviceProperties.maxGridSize[0];
//	}
//
//	printf("Running %d blocks with %d threads/block over %lu elements\n",
//		blocksPerGrid, threadsPerBlock, N);
//
//	floatSample = (float *)malloc(sizeof(float) * sampleLength);
//	doubleSample = (double *)malloc(sizeof(double) * sampleLength);
//
//	for (i = 0; i < nRuns; i++)
//	{
//		long toDeviceTime, kernelTime, fromDeviceTime;
//
//		run_float_test(N, nKernelIters, blocksPerGrid, threadsPerBlock,
//			&toDeviceTime, &kernelTime, &fromDeviceTime,
//			floatSample, sampleLength);
//		meanFloatToDeviceTime += toDeviceTime;
//		meanFloatKernelTime += kernelTime;
//		meanFloatFromDeviceTime += fromDeviceTime;
//
//		run_double_test(N, nKernelIters, blocksPerGrid, threadsPerBlock,
//			&toDeviceTime, &kernelTime, &fromDeviceTime,
//			doubleSample, sampleLength);
//		meanDoubleToDeviceTime += toDeviceTime;
//		meanDoubleKernelTime += kernelTime;
//		meanDoubleFromDeviceTime += fromDeviceTime;
//	}
//
//	meanFloatToDeviceTime /= nRuns;
//	meanFloatKernelTime /= nRuns;
//	meanFloatFromDeviceTime /= nRuns;
//	meanDoubleToDeviceTime /= nRuns;
//	meanDoubleKernelTime /= nRuns;
//	meanDoubleFromDeviceTime /= nRuns;
//
//	meanFloatToDeviceTime /= CLOCKS_PER_SEC;
//	meanFloatKernelTime /= CLOCKS_PER_SEC;
//	meanFloatFromDeviceTime /= CLOCKS_PER_SEC;
//	meanDoubleToDeviceTime /= CLOCKS_PER_SEC;
//	meanDoubleKernelTime /= CLOCKS_PER_SEC;
//	meanDoubleFromDeviceTime /= CLOCKS_PER_SEC;
//
//	printf("For single-precision floating point, mean times for:\n");
//	printf("  Copy to device:   %f s\n", meanFloatToDeviceTime);
//	printf("  Kernel execution: %f s\n", meanFloatKernelTime);
//	printf("  Copy from device: %f s\n", meanFloatFromDeviceTime);
//	printf("For double-precision floating point, mean times for:\n");
//	printf("  Copy to device:   %f s (%.2fx slower than single-precision)\n",
//		meanDoubleToDeviceTime,
//		meanDoubleToDeviceTime / meanFloatToDeviceTime);
//	printf("  Kernel execution: %f s (%.2fx slower than single-precision)\n",
//		meanDoubleKernelTime,
//		meanDoubleKernelTime / meanFloatKernelTime);
//	printf("  Copy from device: %f s (%.2fx slower than single-precision)\n",
//		meanDoubleFromDeviceTime,
//		meanDoubleFromDeviceTime / meanFloatFromDeviceTime);
//
//	return 0;
//}
