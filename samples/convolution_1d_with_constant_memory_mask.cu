
#include <hip/hip_runtime.h>
//#include "cuda_runtime.h"
//#include "device_launch_parameters.h"
//
//#include <iostream>
//
//#ifndef MAX_MASK_WIDTH
//#define MAX_MASK_WIDTH 10
//__constant__ float MASK[MAX_MASK_WIDTH];
//#endif
////this kernel is example for unefficent memory access. 
////this implementation considers only 1D grid. and assumes that mask width is odd number
////We are going to use constant memory to store the mask
//__global__ void convolution_1d_dram_and_constant(float * input, float* output, int array_lenght, int mask_width)
//{
//	int thread_index = blockIdx.x*blockDim.x + threadIdx.x;
//	float temp_value = 0;
//
//	int offset = thread_index - mask_width / 2;
//	for (int i = 0; i < mask_width; i++)
//	{
//		if ((offset + i) >= 0 && (offset + i) < array_lenght)
//		{
//			temp_value += input[offset + i] * MASK[i];
//		}
//	}
//
//	output[thread_index] = temp_value;
//}
//
//void run_code_convolution_2()
//{
//	int array_lenght = 128 * 2;
//	int mask_width = 5;
//
//	int array_byte_size = sizeof(float)*array_lenght;
//	int mask_byte_size = sizeof(float)*mask_width;
//
//	float *h_input_array, *h_mask, *h_output;
//	float *d_input_array, *d_output;
//
//	//host memory allocation
//	h_input_array = (float*)malloc(array_byte_size);
//	h_output = (float*)malloc(array_byte_size);
//	h_mask = (float*)malloc(mask_byte_size);
//
//	//initialize array
//	for (int i = 0; i < array_lenght; i++)
//	{
//		h_input_array[i] = 1.0f;
//	}
//
//	//initialize mask
//	for (int i = 0; i < mask_width; i++)
//	{
//		h_mask[i] = 1.0f;
//	}
//
//	dim3 grid(32);
//	dim3 block((array_lenght) / grid.x);
//
//	//device memory allocation
//	cudaMalloc((float**)&d_input_array, array_byte_size);
//	cudaMalloc((float**)&d_output, array_byte_size);
//
//	//transfer the initiazed arrays to device
//	cudaMemcpy(d_input_array, h_input_array, array_byte_size, cudaMemcpyHostToDevice);
//	cudaMemcpyToSymbol(MASK, h_mask, mask_byte_size);
//
//	//kernel launch
//	convolution_1d_dram_and_constant << <grid, block >> > (d_input_array, d_output, array_lenght, mask_width);
//	//test_kernel << <grid,block >>> (d_input_array);
//	cudaDeviceSynchronize();
//
//	//copy the output back to the host
//	cudaMemcpy(h_output, d_output, array_byte_size, cudaMemcpyDeviceToHost);
//
//	//free the device memory
//	cudaFree(d_input_array);
//	cudaFree(d_output);
//
//	//free the host memory
//	free(h_input_array);
//	free(h_output);
//	free(h_mask);
//}
//
////int main()
////{
////	run_code_convolution_2();
////	system("pause");
////	return 0;
////}