#include "hip/hip_runtime.h"
 #include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include "common.h"

__global__ void sumArrays(int *A, int *B, int *C, const int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N) C[i] = A[i] + B[i];
}

__global__ void sumArraysZeroCopy(int *A, int *B, int *C, const int N)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i < N) C[i] = A[i] + B[i];
}

//int main(int argc, char **argv)
//{
//	// set up device
//	int dev = 0;
//	hipSetDevice(dev);
//
//	// get device properties
//	hipDeviceProp_t deviceProp;
//	hipGetDeviceProperties(&deviceProp, dev);
//
//	// check if support mapped memory
//	if (!deviceProp.canMapHostMemory)
//	{
//		printf("Device %d does not support mapping CPU host memory!\n", dev);
//		hipDeviceReset();
//		exit(EXIT_SUCCESS);
//	}
//
//	// set up data size of vectors
//	int power = 22;
//
//	if (argc > 1) power = atoi(argv[1]);
//
//	int nElem = 1 << power;
//	size_t nBytes = nElem * sizeof(int);
//
//	// part 1: using device memory
//	// malloc host memory
//	int *h_A, *h_B, *hostRef, *gpuRef;
//	h_A = (int *)malloc(nBytes);
//	h_B = (int *)malloc(nBytes);
//	hostRef = (int *)malloc(nBytes);
//	gpuRef = (int *)malloc(nBytes);
//
//	// initialize data at host side
//	initialize(h_A, nElem,INIT_ONE_TO_TEN);
//	initialize(h_B, nElem);
//	memset(gpuRef, 0, nBytes);
//
//	// malloc device global memory
//	int *d_A, *d_B, *d_C;
//	hipMalloc((int**)&d_A, nBytes);
//	hipMalloc((int**)&d_B, nBytes);
//	hipMalloc((int**)&d_C, nBytes);
//
//	// transfer data from host to device
//	hipMemcpy(d_A, h_A, nBytes, hipMemcpyHostToDevice);
//	hipMemcpy(d_B, h_B, nBytes, hipMemcpyHostToDevice);
//
//	// set up execution configuration
//	int iLen = 512;
//	dim3 block(iLen);
//	dim3 grid((nElem + block.x - 1) / block.x);
//
//	sumArrays << <grid, block >> >(d_A, d_B, d_C, nElem);
//	
//	hipDeviceSynchronize();
//	// copy kernel result back to host side
//	hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);
//
//	// free device global memory
//	hipFree(d_A);
//	hipFree(d_B);
//
//	// free host memory
//	free(h_A);
//	free(h_B);
//
//	// part 2: using zerocopy memory for array A and B
//	// allocate zerocpy memory
//	hipHostAlloc((void **)&h_A, nBytes, hipHostMallocMapped);
//	hipHostAlloc((void **)&h_B, nBytes, hipHostMallocMapped);
//
//	// initialize data at host side
//	initialize(h_A, nElem, INIT_ONE_TO_TEN);
//	initialize(h_B, nElem, INIT_ONE_TO_TEN);
//	memset(gpuRef, 0, nBytes);
//
//	// get the mapped device pointer
//	hipHostGetDevicePointer((void **)&d_A, (void *)h_A, 0);
//	hipHostGetDevicePointer((void **)&d_B, (void *)h_B, 0);
//
//	// execute kernel with zero copy memory
//	sumArraysZeroCopy << <grid, block >> >(d_A, d_B, d_C, nElem);
//
//	hipDeviceSynchronize();
//
//	// copy kernel result back to host side
//	hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);
//
//	// free  memory
//	hipFree(d_C);
//	hipHostFree(h_A);
//	hipHostFree(h_B);
//
//	free(hostRef);
//	free(gpuRef);
//
//	// reset device
//	hipDeviceReset();
//	return EXIT_SUCCESS;
//}
