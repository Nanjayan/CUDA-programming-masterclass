#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

//int main(int argc, char **argv) 
//{   								
//	// memory size   128 MBs
//	int isize = 1<<25;   
//	int nbytes = isize * sizeof(float);
//											
//	// allocate the host memory   
//	//float *h_a = (float *)malloc(nbytes);
//	float *h_a;
//	cudaMallocHost((float **)&h_a, nbytes);
//
//	// allocate the device memory   
//	float *d_a; 
//    cudaMalloc((float **)&d_a, nbytes);
//									
//	// initialize the host memory   
//	for(int i=0;i<isize;i++) 
//		h_a[i] = 7;
//									
//	// transfer data from the host to the device   
//	cudaMemcpy(d_a, h_a, nbytes, cudaMemcpyHostToDevice);
//									
//	// transfer data from the device to the host   
//	cudaMemcpy(h_a, d_a, nbytes, cudaMemcpyDeviceToHost);
//							
//	// free memory   
//	cudaFree(d_a);
//	//free(h_a);
//	cudaFreeHost(h_a);
//									
//	// reset device    
//	cudaDeviceReset();   
//	return EXIT_SUCCESS;
//}