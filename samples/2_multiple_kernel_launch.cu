#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""

#include "common.h"

__global__ void simple_kernel()
{
	printf("hello from the kernel \n");
}

//int main(int argc, char ** argv)
//{
//	int dev = 0;
//	hipDeviceProp_t deviceProp;
//	hipGetDeviceProperties(&deviceProp, dev);
//
//	if (deviceProp.concurrentKernels == 0)
//	{
//		printf("> GPU does not support concurrent kernel execution \n");
//		printf("kernel execution will be serialized \n");
//	}
//
//	hipStream_t str1, str2, str3;
//
//	hipStreamCreate(&str1);
//	hipStreamCreate(&str2);
//	hipStreamCreate(&str3);
//
//	simple_kernel << <1, 1, 0, str1 >> > ();
//	simple_kernel << <1, 1, 0, str2 >> > ();
//	simple_kernel << <1, 1, 0, str3 >> > ();
//
//	hipStreamDestroy(str1);
//	hipStreamDestroy(str2);
//	hipStreamDestroy(str3);
//
//	hipDeviceSynchronize();
//	hipDeviceReset();
//	return 0;
//}