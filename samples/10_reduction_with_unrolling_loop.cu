#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "common.h"
#include "cuda_common.cuh"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void reduction_unrolling_blocks2(int * input, int * temp, int size)
{
	int tid = threadIdx.x;

	int BLOCK_OFFSET = blockIdx.x * blockDim.x * 2;

	int index = BLOCK_OFFSET + tid;

	int * i_data = input + BLOCK_OFFSET;

	if ((index + blockDim.x) < size)
	{
		input[index] += input[index + blockDim.x];
	}

	__syncthreads();

	for (int offset = blockDim.x / 2; offset > 0;offset = offset / 2)
	{
		if (tid < offset)
		{
			i_data[tid] += i_data[tid + offset];
		}

		__syncthreads();
	}

	if (tid == 0)
	{
		temp[blockIdx.x] = i_data[0];
	}
}

__global__ void reduction_unrolling_blocks4(int * input, int * temp, int size)
{
	int tid = threadIdx.x;

	int BLOCK_OFFSET = blockIdx.x * blockDim.x * 4;

	int index = BLOCK_OFFSET + tid;

	int * i_data = input + BLOCK_OFFSET;

	if ((index + 3 * blockDim.x) < size)
	{
		int a1 = input[index];
		int a2 = input[index + blockDim.x];
		int a3 = input[index+ 2* blockDim.x];
		int a4 = input[index+ 3 *blockDim.x];
		input[index] = a1 + a2 + a3 + a4;
	}

	__syncthreads();

	for (int offset = blockDim.x / 2; offset > 0; offset = offset / 2)
	{
		if (tid < offset)
		{
			i_data[tid] += i_data[tid + offset];
		}

		__syncthreads();
	}

	if (tid == 0)
	{
		temp[blockIdx.x] = i_data[0];
	}
}


//int main(int argc, char ** argv)
//{
//	printf("Running parallel reduction with unrolling blocks8 kernel \n");
//
//	int size = 1 << 27;
//	int byte_size = size * sizeof(int);
//	int block_size = 128;
//
//	int * h_input, *h_ref;
//	h_input = (int*)malloc(byte_size);
//
//	initialize(h_input, size, INIT_RANDOM);
//
//	int cpu_result = reduction_cpu(h_input, size);
//
//	dim3 block(block_size);
//	dim3 grid((size / block_size) / 2);
//
//	printf("Kernel launch parameters || grid : %d, block : %d \n", grid.x, block.x);
//
//	int temp_array_byte_size = sizeof(int)* grid.x;
//
//	h_ref = (int*)malloc(temp_array_byte_size);
//
//	int * d_input, *d_temp;
//	gpuErrchk(hipMalloc((void**)&d_input, byte_size));
//	gpuErrchk(hipMalloc((void**)&d_temp, temp_array_byte_size));
//
//	gpuErrchk(hipMemset(d_temp, 0, temp_array_byte_size));
//	gpuErrchk(hipMemcpy(d_input, h_input, byte_size,
//		hipMemcpyHostToDevice));
//
//	reduction_unrolling_blocks2 << < grid, block >> > (d_input, d_temp, size);
//
//	gpuErrchk(hipDeviceSynchronize());
//	gpuErrchk(hipMemcpy(h_ref, d_temp, temp_array_byte_size, hipMemcpyDeviceToHost));
//
//	int gpu_result = 0;
//	for (int i = 0; i < grid.x; i++)
//	{
//		gpu_result += h_ref[i];
//	}
//
//	compare_results(gpu_result, cpu_result);
//
//	gpuErrchk(hipFree(d_input));
//	gpuErrchk(hipFree(d_temp));
//	free(h_input);
//	free(h_ref);
//
//	gpuErrchk(hipDeviceReset());
//	return 0;
//}