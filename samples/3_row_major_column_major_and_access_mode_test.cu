#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "hip/hip_runtime.h"
#include ""

#include "common.h"

#define BDIMX 32
#define BDIMY 32

__global__ void setRowReadCol(int * out)
{
	__shared__ int tile[BDIMY][BDIMX];

	int idx = threadIdx.y * blockDim.x + threadIdx.x;

	//store to the shared memory
	tile[threadIdx.y][threadIdx.x] = idx;

	//waiting for all the threads in thread block to reach this point
	__syncthreads();

	//load from shared memory
	out[idx] = tile[threadIdx.x][threadIdx.y];
}

__global__ void setColReadRow(int * out)
{
	__shared__ int tile[BDIMY][BDIMX];

	int idx = threadIdx.y * blockDim.x + threadIdx.x;

	//store to the shared memory
	tile[threadIdx.x][threadIdx.y] = idx;

	//waiting for all the threads in thread block to reach this point
	__syncthreads();

	//load from shared memory
	out[idx] = tile[threadIdx.y][threadIdx.x];
}

__global__ void setRowReadRow(int * out)
{
	__shared__ int tile[BDIMY][BDIMX];

	int idx = threadIdx.y * blockDim.x + threadIdx.x;

	//store to the shared memory
	tile[threadIdx.y][threadIdx.x] = idx;

	//waiting for all the threads in thread block to reach this point
	__syncthreads();

	//load from shared memory
	out[idx] = tile[threadIdx.y][threadIdx.x];
}

//int main(int argc, char **argv)
//{
//	int memconfig = 0;
//	if (argc > 1)
//	{
//		memconfig = atoi(argv[1]);
//	}
//
//
//	if (memconfig == 1)
//	{
//		hipDeviceSetSharedMemConfig(hipSharedMemBankSizeEightByte);
//	}
//	else
//	{
//		hipDeviceSetSharedMemConfig(hipSharedMemBankSizeFourByte);
//	}
//
//	
//	hipSharedMemConfig pConfig;
//	hipDeviceGetSharedMemConfig(&pConfig);
//	printf("with Bank Mode:%s ", pConfig == 1 ? "4-Byte" : "8-Byte");
//	
//
//	// set up array size 2048
//	int nx = BDIMX;
//	int ny = BDIMY;
//
//	bool iprintf = 0;
//	
//	if (argc > 2) iprintf = atoi(argv[1]);
//
//	size_t nBytes = nx * ny * sizeof(int);
//
//	// execution configuration
//	dim3 block(BDIMX, BDIMY);
//	dim3 grid(1, 1);
//	printf("<<< grid (%d,%d) block (%d,%d)>>>\n", grid.x, grid.y, block.x,
//		block.y);
//
//	// allocate device memory
//	int *d_C;
//	hipMalloc((int**)&d_C, nBytes);
//	int *gpuRef = (int *)malloc(nBytes);
//
//	hipMemset(d_C, 0, nBytes);
//	setColReadRow << <grid, block >> >(d_C);
//	hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);
//
//	if (iprintf)  printData("set col read col   ", gpuRef, nx * ny);
//
//	hipMemset(d_C, 0, nBytes);
//	setRowReadRow << <grid, block >> >(d_C);
//	hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);
//
//	if (iprintf)  printData("set row read row   ", gpuRef, nx * ny);
//
//	hipMemset(d_C, 0, nBytes);
//	setRowReadCol << <grid, block >> >(d_C);
//	hipMemcpy(gpuRef, d_C, nBytes, hipMemcpyDeviceToHost);
//
//	if (iprintf)  printData("set row read col   ", gpuRef, nx * ny);
//
//	// free host and device memory
//	hipFree(d_C);
//	free(gpuRef);
//
//	// reset device
//	hipDeviceReset();
//	return EXIT_SUCCESS;
//}
