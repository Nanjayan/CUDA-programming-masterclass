#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""
#include "common.h"
#include "cuda_common.cuh"

__global__ void k1()
{
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	if (gid == 0)
	{
		printf("This is a test 1 \n");
	}
}

//int main(int argc, char ** argv)
//{
//	int size = 1 << 15;
//
//	hipStream_t stm1,stm2,stm3;
//	hipStreamCreate(&stm1);
//	hipStreamCreate(&stm2);
//	hipStreamCreate(&stm3);
//
//	hipEvent_t event1;
//	hipEventCreateWithFlags(&event1, hipEventDisableTiming);
//
//	dim3 block(128);
//	dim3 grid(size / block.x);
//	
//	k1 << <grid, block, 0, stm1 >> > ();
//	hipEventRecord(event1, stm1);
//	hipStreamWaitEvent(stm3, event1, 0);
//
//	k1 << <grid, block, 0, stm2 >> > ();
//	k1 << <grid, block, 0, stm3 >> > ();
//
//	hipEventDestroy(event1);
//
//	hipStreamDestroy(stm1);
//	hipStreamDestroy(stm2);
//	hipStreamDestroy(stm3);
//
//	hipDeviceSynchronize();
//
//	hipDeviceReset();
//	return 0;
//}
