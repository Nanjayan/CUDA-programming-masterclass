#include "cuda_common.cuh"

//void query_device()
//{
//	int iDev = 0;
//	hipDeviceProp_t iProp;
//	hipGetDeviceProperties(&iProp, iDev);
//
//	printf("Device %d: %s\n", iDev, iProp.name);
//	printf("  Number of multiprocessors:                     %d\n",
//		iProp.multiProcessorCount);
//	//printf("  Number of multiprocessors:                     %d\n",
//	//	iProp.);
//	printf("  Compute capability       :                     %d.%d\n",
//		iProp.major,iProp.minor);
//	printf("  Total amount of global memory:                 %4.2f KB\n",
//		iProp.totalGlobalMem/ 1024.0);
//	printf("  Total amount of constant memory:               %4.2f KB\n",
//		iProp.totalConstMem / 1024.0);
//	printf("  Total amount of shared memory per block:       %4.2f KB\n",
//		iProp.sharedMemPerBlock / 1024.0);
//	printf("  Total amount of shared memory per MP:          %4.2f KB\n",
//		iProp.sharedMemPerMultiprocessor / 1024.0);
//	printf("  Total number of registers available per block: %d\n",
//		iProp.regsPerBlock);
//	printf("  Warp size:                                     %d\n",
//		iProp.warpSize);
//	printf("  Maximum number of threads per block:           %d\n",
//		iProp.maxThreadsPerBlock);
//	printf("  Maximum number of threads per multiprocessor:  %d\n",
//		iProp.maxThreadsPerMultiProcessor);
//	printf("  Maximum number of warps per multiprocessor:    %d\n",
//		iProp.maxThreadsPerMultiProcessor / 32);
//	printf("  Maximum Grid size                         :    (%d,%d,%d)\n",
//		iProp.maxGridSize[0], iProp.maxGridSize[1], iProp.maxGridSize[2]);
//	printf("  Maximum block dimension                   :    (%d,%d,%d)\n",
//		iProp.maxThreadsDim[0], iProp.maxThreadsDim[1], iProp.maxThreadsDim[2]);
//}