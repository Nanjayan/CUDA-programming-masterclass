#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include "common.h"
#include "cuda_common.cuh"

#define BLOCK_SIZE 64

__global__ void scan_for_compact(int * input, int * output_index_array,int* auxiliry_array, int input_size)
{
	int idx = threadIdx.x;
	int gid = blockDim.x*blockIdx.x + threadIdx.x;

	__shared__ int local_input[BLOCK_SIZE];

	if (input[gid] >0)
	{
		local_input[idx] = 1;
	}
	else
	{
		local_input[idx] = 0;
	}

	__syncthreads();

	// reduction phase
	// this can be optimized check wether global memory access for "d" or calculation here is better
	int d = ceilf(log2f(BLOCK_SIZE));
	int denominator = 0;
	int offset = 0;
	//reduction should happen per block
	for (int i = 1; i <= d; i++)
	{
		denominator = 1 << i;
		offset = 1 << (i - 1);
		if (((idx + 1) % denominator) == 0)
		{
			local_input[idx] += local_input[idx - offset];
		}
		__syncthreads();
	}

	////end of reduction phase

	//// start of  down-sweep phase
	if (idx == (BLOCK_SIZE - 1))
	{
		local_input[idx] = 0;
	}

	int temp = 0;
	int sawp_aux = 0;

	for (int i = d; i > 0; i--)
	{
		temp = 1 << i;
		if ((idx != 0) && (idx + 1) % temp == 0)
		{
			sawp_aux = local_input[idx];
			local_input[idx] += local_input[idx - (temp / 2)];
			local_input[idx - (temp / 2)] = sawp_aux;
		}
		__syncthreads();
	}

	//can this be add to if condition at the begining of the down sweep phase 
	if (idx == (BLOCK_SIZE - 1))
	{
		auxiliry_array[blockIdx.x] = local_input[idx];
		//printf("%d \n", auxiliry_array[blockIdx.x]);
	}
	output_index_array[gid] = local_input[idx];
}

__global__ void scan_summation_for_compact(int * output_index_array, int * auxiliry_array, int input_size)
{
	int idx = threadIdx.x;
	int gid = blockDim.x*blockIdx.x + threadIdx.x;

	__shared__ int local_input[BLOCK_SIZE];

	local_input[idx] = output_index_array[gid];
	__syncthreads();

	for (int i = 0; i < blockIdx.x; i++)
	{
		local_input[idx] += auxiliry_array[i];
	}

	output_index_array[gid] = local_input[idx];
	//printf("gid : %d, value : %d \n", gid, output_index_array[gid]);
}

__global__ void compact_1D_array( int * input, int * output, int * output_index_array, int array_size)
{
	int gid = blockDim.x*blockIdx.x + threadIdx.x;
	
	//TO DO handle when gid ==0
	//this is very unefficient in memory management
	if (gid > 0 && gid < array_size)
	{
		if (output_index_array[gid] != output_index_array[gid - 1])
		{
			//printf("gid : %d , index :%d , value : %d, prev_value : %d \n",gid, output_index_array[gid], input[gid], input[gid-1]);
			output[output_index_array[gid]] = input[gid-1];
		}
	}
}

void run_compact()
{
	int input_size = 1 << 7;
	int input_byte_size = input_size * sizeof(int);
	dim3 block(BLOCK_SIZE);
	dim3 grid(input_size / block.x);
	int aux_byte_size = sizeof(int)*grid.x;

	int* h_input, *h_ref, *h_aux_ref, *h_output;
	h_input = (int*)malloc(input_byte_size);
	h_ref = (int*)malloc(input_byte_size);
	h_aux_ref = (int*)malloc(aux_byte_size);

	for (int i = 0; i < input_size; i++)
	{
		if (i % 5 == 0)
		{
			h_input[i] = i;
		}
		else
		{
			h_input[i] = 0;
		}
	}

	int * d_input, *d_output_index_array, *d_aux, *d_sum_input, *d_sum_aux, *d_output;
	gpuErrchk(hipMalloc((int**)&d_input, input_byte_size));
	gpuErrchk(hipMalloc((int**)&d_output_index_array, input_byte_size));
	gpuErrchk(hipMalloc((int**)&d_aux, aux_byte_size));
	gpuErrchk(hipMalloc((int**)&d_sum_input, input_byte_size));
	gpuErrchk(hipMalloc((int**)&d_sum_aux, aux_byte_size));

	gpuErrchk(hipMemcpy(d_input, h_input, input_byte_size, hipMemcpyHostToDevice));
	scan_for_compact << <grid, block >> > (d_input, d_output_index_array,d_aux, input_size);
	gpuErrchk(hipDeviceSynchronize());

	//gpuErrchk(hipMemcpy(d_sum_input, d_output_index_array, input_byte_size, hipMemcpyDeviceToDevice));
	gpuErrchk(hipMemcpy(d_sum_aux, d_aux, aux_byte_size, hipMemcpyDeviceToDevice));
	scan_summation_for_compact << <grid, block >> > (d_output_index_array, d_sum_aux, input_size);
	gpuErrchk(hipDeviceSynchronize());
	
	gpuErrchk(hipMemcpy(h_ref, d_output_index_array, input_byte_size, hipMemcpyDeviceToHost));

	int compact_output_size = h_ref[input_size - 1];
	int compact_output_byte_size = sizeof(float)*compact_output_size;

	h_output = (int*)malloc(compact_output_byte_size);
	gpuErrchk(hipMalloc((int**)&d_output, compact_output_byte_size));
	
	compact_1D_array << <grid, block >> > (d_input, d_output, d_output_index_array, input_size);
	gpuErrchk(hipDeviceSynchronize());

	gpuErrchk(hipMemcpy(h_output, d_output, compact_output_byte_size, hipMemcpyDeviceToHost));

	for (int i=0;i<compact_output_size;i++)
	{
		printf("%d \n",h_output[i]);
	}
	
	hipFree(d_sum_input);
	hipFree(d_sum_aux);
	hipFree(d_input);
	hipFree(d_aux);

	free(h_input);
	free(h_aux_ref);
	free(h_ref);
}

//int main()
//{
//	run_compact();
//	system("pause");
//	return 0;
//}