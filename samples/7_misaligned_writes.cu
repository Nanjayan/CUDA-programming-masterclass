#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "hip/hip_runtime.h"



__global__ void misaligned_write_test(float* a, float* b, float *c, int size, int offset)
{
	int gid = blockIdx.x * blockDim.x + threadIdx.x;
	int k = gid + offset;

	if (k < size)
		c[k] = a[gid] + b[gid];
}

//int main(int argc, char** argv)
//{
//	printf("Runing 1D grid \n");
//	int size = 1 << 25;
//	int block_size = 128;
//	unsigned int byte_size = size * sizeof(float);
//	int offset = 0;
//
//	if (argc > 1)
//		offset = atoi(argv[1]);
//
//	printf("Input size : %d \n", size);
//
//	float * h_a, *h_b, *h_ref;
//	h_a = (float*)malloc(byte_size);
//	h_b = (float*)malloc(byte_size);
//	h_ref = (float*)malloc(byte_size);
//
//
//	if (!h_a)
//		printf("host memory allocation error \n");
//
//	for (size_t i = 0; i < size; i++)
//	{
//		h_a[i] = i % 10;
//		h_b[i] = i % 7;
//	}
//
//	dim3 block(block_size);
//	dim3 grid((size + block.x - 1) / block.x);
//
//	printf("Kernel is lauch with grid(%d,%d,%d) and block(%d,%d,%d) \n",
//		grid.x, grid.y, grid.z, block.x, block.y, block.z);
//
//	float *d_a, *d_b, *d_c;
//
//	cudaMalloc((void**)&d_a, byte_size);
//	cudaMalloc((void**)&d_b, byte_size);
//	cudaMalloc((void**)&d_c, byte_size);
//	cudaMemset(d_c, 0, byte_size);
//
//	cudaMemcpy(d_a, h_a, byte_size, cudaMemcpyHostToDevice);
//	cudaMemcpy(d_b, h_b, byte_size, cudaMemcpyHostToDevice);
//
//	misaligned_write_test << <grid, block >> > (d_a, d_b, d_c, size, offset);
//
//	cudaDeviceSynchronize();
//	cudaMemcpy(h_ref, d_c, byte_size, cudaMemcpyDeviceToHost);
//
//	cudaFree(d_c);
//	cudaFree(d_b);
//	cudaFree(d_a);
//	free(h_ref);
//	free(h_b);
//	free(h_a);
//}