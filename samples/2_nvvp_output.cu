#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"
#include ""

#include "common.h"

__global__ void stream_test(int* in, int * out, int size)
{
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	if (gid < size)
	{
		//THIS FOR LOOP IS ONLY FOR VISUALIZING PURPOSE  
		for (int  i = 0; i < 25; i++)
		{
			out[gid] = in[gid] + (in[gid] - 1) * (gid % 10);
		}
	}
}

//int main(int argc, char ** argv)
//{
//	int size = 1 << 18;
//	int byte_size = size * sizeof(int);
//
//	//initialize host pointer
//	int* h_in, *h_ref;
//	h_in = (int *)malloc(byte_size);
//	h_ref = (int *)malloc(byte_size);
//	initialize(h_in,INIT_RANDOM);
//
//	//allocate device pointers
//	int * d_in, *d_out;
//	hipMalloc((void**)&d_in, byte_size);
//	hipMalloc((void**)&d_out, byte_size);
//
//	//transfer data from host to device
//	hipMemcpy(d_in, h_in, byte_size, hipMemcpyHostToDevice);
//	
//	//kernel launch
//	dim3 block(128);
//	dim3 grid(size / block.x);
//	
//	stream_test << <grid, block >>> (d_in,d_out, size);
//	hipDeviceSynchronize();
//
//	//copy the memory back to host
//	hipMemcpy(h_ref, d_out, byte_size, hipMemcpyDeviceToHost);
//
//	hipDeviceReset();
//	return 0;
//}