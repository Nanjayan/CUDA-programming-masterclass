#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>

#include "common.h"
#include "cuda_common.cuh"

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

__global__ void reduction_kernel_complete_unrolling(int * int_array,
	int * temp_array, int size)
{
	int tid = threadIdx.x;

	//element index for this thread
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	//local data pointer
	int * i_data = int_array + blockDim.x * blockIdx.x;

	if (blockDim.x == 1024 && tid < 512)
		i_data[tid] += i_data[tid + 512];
	__syncthreads();

	if (blockDim.x == 512 && tid < 256)
		i_data[tid] += i_data[tid + 256];
	__syncthreads();

	if (blockDim.x == 256 && tid < 128)
		i_data[tid] += i_data[tid + 128];
	__syncthreads();

	if (blockDim.x == 128 && tid < 64)
		i_data[tid] += i_data[tid + 64];
	__syncthreads();

	if (tid < 32)
	{
		volatile int * vsmem = i_data;
		vsmem[tid] += vsmem[tid + 32];
		vsmem[tid] += vsmem[tid + 16];
		vsmem[tid] += vsmem[tid + 8];
		vsmem[tid] += vsmem[tid + 4];
		vsmem[tid] += vsmem[tid + 2];
		vsmem[tid] += vsmem[tid + 1];
	}

	if (tid == 0)
	{
		temp_array[blockIdx.x] = i_data[0];
	}
}

//int main(int argc, char ** argv)
//{
//	printf("Running parallel reduction with complete unrolling kernel \n");
//
//	int size = 1 << 22;
//	int byte_size = size * sizeof(int);
//	int block_size = 128;
//
//	int * h_input, *h_ref;
//	h_input = (int*)malloc(byte_size);
//
//	initialize(h_input, size, INIT_RANDOM);
//
//	int cpu_result = reduction_cpu(h_input, size);
//
//	dim3 block(block_size);
//	dim3 grid(size / block_size);
//
//	printf("Kernel launch parameters || grid : %d, block : %d \n", grid.x, block.x);
//
//	int temp_array_byte_size = sizeof(int)* grid.x;
//
//	h_ref = (int*)malloc(temp_array_byte_size);
//
//	int * d_input, *d_temp;
//	gpuErrchk(hipMalloc((void**)&d_input, byte_size));
//	gpuErrchk(hipMalloc((void**)&d_temp, temp_array_byte_size));
//
//	gpuErrchk(hipMemset(d_temp, 0, temp_array_byte_size));
//	gpuErrchk(hipMemcpy(d_input, h_input, byte_size,
//		hipMemcpyHostToDevice));
//
//	reduction_kernel_complete_unrolling <<< grid, block >> > (d_input, d_temp, size);
//
//	gpuErrchk(hipDeviceSynchronize());
//	gpuErrchk(hipMemcpy(h_ref, d_temp, temp_array_byte_size, hipMemcpyDeviceToHost));
//
//	int gpu_result = 0;
//	for (int i = 0; i < grid.x; i++)
//	{
//		gpu_result += h_ref[i];
//	}
//
//	compare_results(gpu_result, cpu_result);
//
//	gpuErrchk(hipFree(d_input));
//	gpuErrchk(hipFree(d_temp));
//	free(h_input);
//	free(h_ref);
//
//	gpuErrchk(hipDeviceReset());
//	return 0;
//}